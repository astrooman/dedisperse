#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "kernels.cuh"

__global__ void InitDelaysKernel(unsigned int *delays, unsigned int nchans, float dm, float ftop, float foff, float tsamp) {
    tsamp *= 1e+03;     // conver sampling time from seconds to milliseconds
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float fbott = ftop + (float)idx * foff;
    // NOTE: The below calculation is done for delay in milliseconds with frequencies in MHz
    delays[idx] = (int)(4.15e+06 * ( 1.0f / (fbott * fbott) - 1.0f / (ftop * ftop)) * dm / tsamp);
}

__global__ void TransposeKernel(unsigned char *indata, unsigned char *outdata, unsigned int outsamples, unsigned int perthread, unsigned int perblock) {
    unsigned int sampid;
    for (int isamp = 0; isamp < perblock; isamp++) {
        sampid = blockIdx.x * perblock + isamp;
        if (sampid < outsamples) {
            for (int ichan = 0; ichan < perthread; ichan++) {
                outdata[(threadIdx.x * perthread + ichan) * outsamples + (blockIdx.x * perblock + isamp)] = indata[(blockIdx.x * perblock + isamp) * blockDim.x * perthread + threadIdx.x * perthread + ichan];
            }
        }
    }
}

__global__ void DedisperseKernel(unsigned char *indata, float *outdata, unsigned int *delays, unsigned int insamples, unsigned int outsamples, unsigned int perblock, unsigned int perthread) {

    unsigned int chanid;
    unsigned int sampid;

    for (unsigned int isamp = 0; isamp < perblock; isamp++) {
        sampid = blockIdx.x * perblock + isamp;
        if (sampid < outsamples) {
            for (unsigned int ichan = 0; ichan < perthread; ichan ++) {
                chanid = threadIdx.x * perthread + ichan;
                outdata[chanid * outsamples + sampid] = (float)indata[chanid * insamples + sampid + delays[chanid]];
            }
        }
    }

}
